#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <chrono>
#include "helper.h"

constexpr int N = 100000;

using DataTp = int;

DataTp randomNum() {
    std::random_device randomDevice;
    std::mt19937 mt(randomDevice());
    std::uniform_real_distribution<double> distribution(0,10);
    return DataTp (distribution(mt));
}

__global__ void GPU_vectorAdd(const DataTp *vector_1, const DataTp *vector_2, DataTp *result) {

    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N) {
        result[id] = vector_1[id] +vector_2[id];
    }
}

void CPU_vectorAdd(const DataTp *vector_1, const DataTp *vector_2, DataTp *result) {
    for(int i = 0; i < N; i++) {
        result[i] = vector_1[i] + vector_2[i];
    }
}

void getRandomVector(DataTp *vector) {

    for(int i = 0; i < N; i++) {
        vector[i] = randomNum();
    }
}

void vectorPrint(DataTp *vector) {

    for(int i = 0; i < N; i++) {
        std::cout<<vector[i]<<"\t";
    }
    std::cout<<std::endl;
}

//void test_cpu(DataTp *vector_1, DataTp *vector_2, DataTp *result) {
//    getRandomVector(vector_1);
//    getRandomVector(vector_2);
//    CPU_vectorAdd(vector_1,vector_2,result);
//    vectorPrint(vector_1);
//    vectorPrint(vector_2);
//    vectorPrint(result);
//}


int main() {

    auto * vector_1 = new DataTp[N];
    auto * vector_2 = new DataTp[N];
    auto * CPU_result = new DataTp[N];
    auto * GPU_result = new DataTp[N];

    //test_cpu(vector_1,vector_2,CPU_result);
    DataTp *cuda_v01,*cuda_v02,*cuda_result;

    getRandomVector(vector_1);
    getRandomVector(vector_2);

//    TI_CPU(CPU)
//    CPU_vectorAdd(vector_1,vector_2,CPU_result);
//    TO_CPU(CPU,"cpu takes: ",1)

    hipMalloc((void**)&cuda_v01,N*sizeof(DataTp));
    hipMalloc((void**)&cuda_v02,N*sizeof(DataTp));
    hipMalloc((void**)&cuda_result,N*sizeof(DataTp));

    hipMemcpy(cuda_v01,vector_1,N* sizeof(DataTp),hipMemcpyHostToDevice);
    hipMemcpy(cuda_v02,vector_2,N* sizeof(DataTp),hipMemcpyHostToDevice);

//    TI_GPU(GPU1)
//    GPU_vectorAdd<<<N,1>>>(cuda_v01,cuda_v02,cuda_result);
//    TO_GPU(GPU1,"time",1)


    constexpr int threadNX = 32;
    constexpr int threadNY =  4;
    dim3 block(threadNX, threadNY);
    int Nx_blocks = (N  + threadNX - 1 ) / threadNX;
    int Ny_blocks = (1  + threadNY - 1 ) / threadNY;
    dim3 grid(Nx_blocks, Ny_blocks);

    TI_GPU(GPU2)
    for (int i =0; i<10000; i++) {
        GPU_vectorAdd<<<grid, block>>>(cuda_v01, cuda_v02, cuda_result);
    }TO_GPU(GPU2,"time",10000)

    hipMemcpy(GPU_result, cuda_result, N * sizeof(DataTp), hipMemcpyDeviceToHost);

    //vectorPrint(vector_1);
    //vectorPrint(vector_2);

//    std::cout<<"CPU result:"<<std::endl;
//    vectorPrint(CPU_result);
//    std::cout<<"GPU result:"<<std::endl;
   //vectorPrint(GPU_result);

    hipFree(cuda_v01);
    hipFree(cuda_v02);
    hipFree(cuda_result);
    return 0;
}
